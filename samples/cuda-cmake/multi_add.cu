#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "multi_add.h"

__global__
void MultiAddCuda(double *a, double *b, double *c, double *d, unsigned int n) {
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < n) {
		for (int loop = 0; loop < 100'000; ++loop)
			d[index] = a[index] * b[index] + c[index];
	}
}


void MultiAdd(double *a, double *b, double *c, double *d, unsigned int n) {
	double *cua, *cub, *cuc, *cud;

	hipMalloc(&cua, n*sizeof(double));
	hipMalloc(&cub, n*sizeof(double));
	hipMalloc(&cuc, n*sizeof(double));
	hipMalloc(&cud, n*sizeof(double));

	hipMemcpy(cua, a, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cub, b, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cuc, c, n*sizeof(double), hipMemcpyHostToDevice);

	// 把循环放在这里是为了避免多次重复申请、释放显存空间
	MultiAddCuda<<<(n+255)/256, 256>>>(cua, cub, cuc, cud, n);

	hipMemcpy(d, cud, n*sizeof(double), hipMemcpyDeviceToHost);

	hipFree(cua);
	hipFree(cub);
	hipFree(cuc);
	hipFree(cud);
}

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "./sharedlib.h"

__global__ void cuadd(double* a, double* b, double* c, unsigned int N){
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N){
        c[id] = a[id] + b[id];
    }
}

void vecadd(double* a, double* b, double* c, unsigned int N){
    double* cua;
    double* cub;
    double* cuc;
    hipMalloc(&cua, N*sizeof(double));
    hipMalloc(&cub, N*sizeof(double));
    hipMalloc(&cuc, N*sizeof(double));
    hipMemcpy(cua, a, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(cub, b, N*sizeof(double), hipMemcpyHostToDevice);
    cuadd<<<(N+127)/128,128>>>(cua,cub,cuc,N);
    hipMemcpy(c, cuc, N*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(cua);
    hipFree(cub);
    hipFree(cuc);
}


